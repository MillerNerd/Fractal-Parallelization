#include <stdlib.h>
#include <stdio.h>
#include <malloc.h>
#include <assert.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define THREADSPERBLOCK 256

#define xMin 0.74395
#define xMax 0.74973
#define yMin 0.11321
#define yMax 0.11899

static __global__ void FractalKernel(int width, int from, int to, int maxdepth, double dx, double dy, unsigned char cnt[])
{
  // kernel code goes in here

  /* compute thread index */
  int index = threadIdx.x + blockIdx.x * blockDim.x + (width * from);
  double cx, cy, x, y, x2, y2;
  int row, col, depth;

  /* compute fractal */
  if(index < (width * to))
  {
    //calculate row and col
    col = index % width;
    row = index / width;

    cy = yMin + row * dy;
    cx = xMin + col * dx;
    x = -cx;
    y = -cy;
    depth = maxdepth;
    do
    {
      x2 = x * x;
      y2 = y * y;
      y = 2 * x * y - cy;
      x = x2 - y2 - cx;
      depth--;
    } while ((depth > 0) && ((x2 + y2) <= 5.0));
    cnt[row * width + col] = depth & 255;
  }
}

extern "C" unsigned char *GPU_Init(int size)
{
  /* device copies */
  unsigned char *d_cnt;

  // allocate array on GPU and return pointer to it
  hipMalloc((void **) &d_cnt, size);

  return d_cnt;
}

extern "C" void GPU_Exec(int width, int from, int to, int maxdepth, double dx, double dy, unsigned char *cnt_d)
{
  // call the kernel (and do nothing else)
  FractalKernel <<< (width * (to - from) + THREADSPERBLOCK - 1) / THREADSPERBLOCK , THREADSPERBLOCK >>> (width, from, to, maxdepth, dx, dy, cnt_d);
}

extern "C" void GPU_Fini(unsigned char *cnt, unsigned char *cnt_d, int size)
{
  // copy the pixel data to the CPU and deallocate the GPU array
  hipMemcpy(cnt, cnt_d, size, hipMemcpyDeviceToHost);
  hipFree(cnt_d);
}
